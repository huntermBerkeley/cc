
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <string>
#include <inttypes.h>
#include <iostream>
#include <vector>
#include <assert.h>
#include <fstream>
//Init cuda here


// __device__ void VecTest(float * A, float* B, size_t n){
//
//   size_t tid = threadIdx.x;
//   if (tid < n){
//     B[N]  = A[N];
//   }
//
//   __syncthreads();
//   return;
// }

typedef unsigned long long int uint64_cu;

#ifndef MAX_VEC
#define MAX_VEC 5000
#endif

//credit to stackoverflow
//https://stackoverflow.com/questions/5447570/cuda-atomic-operations-on-unsigned-chars

using namespace std;

// __device__ static inline char atomicCAS(char* address, char expected, char desired) {
//     size_t long_address_modulo = (size_t) address & 3;
//     auto* base_address = (unsigned int*) ((char*) address - long_address_modulo);
//     unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
//
//     unsigned int sel = selectors[long_address_modulo];
//     unsigned int long_old, long_assumed, long_val, replacement;
//     char old;
//
//     long_val = (unsigned int) desired;
//     long_old = *base_address;
//     do {
//         long_assumed = long_old;
//         replacement =  __byte_perm(long_old, long_val, sel);
//         long_old = atomicCAS(base_address, long_assumed, replacement);
//         old = (char) ((long_old >> (long_address_modulo * 8)) & 0x000000ff);
//     } while (expected == old && long_assumed != long_old);
//
//     return old;
// }

void printrowkern(uint64_t row, char * vec, uint64_t*lengths){

    std::cout << "len: " << lengths[row] << endl;
    std::cout << "[ ";
    for (int j = 0; j < lengths[row]; j++){
      cout << vec[row*MAX_VEC+j];
    }
    std:: cout << " ]" << endl;

}

void printLenskern(std::vector<uint64_t> rows, uint64_t*lengths){

  std::cout << "[ ";
    for (int j = 0; j < rows.size(); j++){
      std::cout << rows.at(j) << ": " << lengths[rows.at(j)] << ", ";
    }
    std:: cout << " ]" << endl;

}

void printCudaVec(uint64_t nnz, uint64_t* cudaVec){

  uint64_t * copy;

  copy = new uint64_t[nnz];

  hipMemcpy(copy, cudaVec,  nnz*sizeof(uint64_t), hipMemcpyDeviceToHost);

  for (uint64_t i =0; i < nnz; i++){
    cout << i << ": [ " << copy[i] << " ]  ";

    if (i % 5 == 4){
      cout << endl;
    }
  }
  cout << endl;

  delete copy;
}

void printCudaStars(uint64_t nnz, bool* cudaVec){

  bool * copy;

  copy = new bool[nnz];

  hipMemcpy(copy, cudaVec,  nnz*sizeof(bool), hipMemcpyDeviceToHost);

  for (uint64_t i =0; i < nnz; i++){
    cout << i << ": [ " << copy[i] << " ]" << endl;
  }

  delete copy;
}

//convert sparse char mat to boolean ints
__global__ void mat_char_to_int(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char* Avals, uint64_t *Bcols, uint64_t * Brows, uint64_t * Bvals){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  Brows[tid] = Arows[tid];
  Bcols[tid] = Acols[tid];
  Bvals[tid] = 1;
}




__global__ void naive_cond_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char * Avals, uint64_t * parent, bool * star){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  uint64_t u = Arows[tid];
  uint64_t v = Acols[tid];

  uint64_t parent_u = parent[u];
  uint64_t parent_v = parent[v];


  //retreive f earlier
  uint64_t gparent_u = parent[parent[u]];
  uint64_t old;

  //star hook procedure
  if (star[u] && parent[u] > parent[v]){
    old = (uint64_t) atomicCAS( (uint64_cu *) parent+parent_u, (uint64_cu) gparent_u, (uint64_cu) parent_v);
    //if this is the case we must have succeeded
    if (old == gparent_u){
      return;
    }
    parent_v = parent[v];
    parent_u = parent[u];
    gparent_u = parent[parent_u];
  }


}

__global__ void parent_cond_hook(uint64_t nnz, uint64_t * parent, uint64_t* gparent, bool * star, char* contigs, uint64_t * contig_lens, char* contigs_holder, uint64_t * contig_lens_holder){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  //for parent cond hook, if I am not a star, set my parent to my grandparent

  //if star[u]
  // parent[u] = parent[parent[u]]
  uint64_t gparent_u = gparent[tid];
  uint64_t parent_u = parent[tid];

  if (star[tid]){

    //absorb from your parent
    //first copy over your material

    //compress parents
    uint64_t my_contig_len = contig_lens[tid];
    uint64_t my_parent_len = contig_lens[parent_u];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_parent = contigs+MAX_VEC*parent_u;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    //copy from my parent
    for (int i =0; i < my_parent_len; i++){
      my_output[i+my_contig_len] = my_parent[i];
    }

    //copy to new len
    contig_lens_holder[tid] = my_contig_len+my_parent_len;

    //and absorb
    parent[tid] = gparent_u;



  } else {

    uint64_t my_contig_len = contig_lens[tid];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    contig_lens_holder[tid] = my_contig_len;

  }
  // if (parent[tid] =  parent[parent[tid]]){
  //   star[tid] = false;
  // }


}


__global__ void naive_uncond_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char * Avals, uint64_t * parent, bool * star){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  uint64_t u = Arows[tid];
  uint64_t v = Acols[tid];

  uint64_t parent_u = parent[u];
  uint64_t parent_v = parent[v];

  //retreive f earlier
  uint64_t gparent_u = parent[parent[u]];
  uint64_t old;

  //star hook procedure
  if (star[u] && parent[u] != parent[v]){
    old = (uint64_t) atomicCAS( (uint64_cu *) parent+parent_u, (uint64_cu) gparent_u, (uint64_cu) parent_v);
    //if this is the case we must have succeeded
    if (old == gparent_u){
      return;
    }
    parent_v = parent[v];
    parent_u = parent[u];
    gparent_u = parent[parent_u];
  }


}

__global__ void shortcutting(uint64_t nnz, uint64_t * parents, uint64_t * gparents, bool * stars){

  //assume gparents already defined
  uint64_t tid = threadIdx.x +  blockIdx.x * blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  uint64_t v = tid;

  //star hook procedure
  if (!stars[v]){

    parents[v] = gparents[v];

  }


}

__global__ void setGrandparents(uint64_t nnz, uint64_t * parents, uint64_t * grandparents){

  //assume gparents already defined
  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  grandparents[tid] = parents[parents[tid]];

  return;

}

__global__ void reset_star(uint64_t nnz, bool * stars){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  stars[tid] = true;


}


//initialize every thread to be it's own parent
__global__ void init_parent(uint64_t nnz,  uint64_t* parent){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    parent[tid] = tid;
  }

}

//initialize lengths to be 0
__global__ void init_contig_lens(uint64_t nnz,  uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    contig_lens[tid] = 0;
  }

}

//assert lengths are null
__global__ void assert_contig_lens(uint64_t nnz,  uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    if (contig_lens[tid] != 0){
      printf("contiig length %llu not 0\n", tid);
    }
  }

}

//initialize lengths to be 0
__global__ void init_contigs(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, char* contigs, uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){

    //grab row
    char my_val = Avals[tid];
    contigs[Arows[tid]*MAX_VEC] = my_val;
    contig_lens[Arows[tid]] = 1;
  }

}



// __device__ char semiring_multiply(char a, char b){
//
//   printf("Multiplying %c, %c\n", a,b);
//   if (a == 'z' ||  b == 'z')
//     return 'z';
//
//   if (a == 0x20) return b;
//
//   return a;
// }

// __device__ char semiring_add(char a, char b){
//
//   printf("adding %c, %c\n", a,b);
//   if (a == 0x20){
//     return b;
//   }
//   if (b == 0x20){
//     return a;
//   }
//   //both nonzero, bad path
//   //this will corrupt any future adds to this index as well
//   return 'z';
// }

__global__ void copy_kernel(double * to_copy, double* items, size_t n) {
  int tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    to_copy[tid] = items[tid];
  }
}

__global__ void uint_copy_kernel(uint64_t* to_fill, uint64_t* to_copy, size_t n) {
  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    to_fill[tid] = to_copy[tid];
  }
}

__global__ void copy_kernel_char(char * to_copy, char* items, size_t n) {
  int tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    printf("Tid %d reporting\n", tid);
    to_copy[tid] = items[tid];
  }
}

//__global__ void kmer_copy_kernel(uint64_t contig_num, char * contigs, )

//After the conditional hooking step, we should push any updated reads into the contigs
//because this happens first, the len of the contigs must be 1
__global__ void update_leads(uint64_t nnz, char * contigs, uint64_t * contig_lens, uint64_t num_updates, char * updates, uint64_t * update_lens, uint64_t * parent){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= num_updates) return;


  uint64_t contig_index = parent[tid];

  assert(contig_lens[contig_index] == 1);

  contig_lens[contig_index] += update_lens[tid];

  //move the first intem back to the last index in preparation for the copy kernel
  //0th index to

  //was a -1 on the left
  //lets split this into parts
  contigs[MAX_VEC*contig_index+contig_lens[contig_index]-1] = contigs[MAX_VEC*contig_index];


  //copy kernel moved from cc
  for (int i = 0; i < contig_lens[contig_index]-1; i++){

    contigs[MAX_VEC*contig_index+i] = updates[MAX_VEC*tid+i];

  }

  //finished>
  return;

}



__global__ void vec_kernel(int nnz, int* vec){
  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;


  vec[tid] = 1;
}



__global__ void clear_kernel(int nnz, char*vec){
  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;


  vec[tid] =0x20;

}




__global__ void check_stars(uint64_t nnz,  bool * stars, int* converged){

    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= nnz) return;

    if (stars[tid]){
      //printf("We should not converge: %llu\n", tid);

      //swap 1 with 0 if it hasn't happened
      //come back and time this
      converged[0] = 0;
      //cas works, let's test regular convergence
      //atomicCAS(converged,1,0);

    }

    return;

}



// void  fill_wrapper(int nnz, int*vals, int*rows, int*cols){
//
//   int blocknums  = (nnz - 1)/ 1024 + 1;
//
//   fill_matrix<<<blocknums, 1024>>>(nnz, vals,rows,cols);
//
// }

void copy_wrapper(double * to_copy, double* items, size_t n){

  copy_kernel<<<1,n>>>(to_copy, items, n);

}

//check if all items in vec are false: if true, converged
bool starConverged(uint64_t nnz, bool*stars){

  int * converged;

  hipMallocManaged((void **)&converged,1*sizeof(int));

  //set to true initially
  converged[0] = 1;

  uint64_t blocknums = (nnz -1)/1024 + 1;

  check_stars<<<blocknums, 1024>>>(nnz, stars, converged);
  hipDeviceSynchronize();

  bool result = true;

  result = (converged[0] == 1);

  std::cout << "Result: " << result << "." << std::endl;
  hipFree(converged);

  return result;

}

void  fill_vector(int nnz, int*vector){

  int blocknums  = (nnz - 1)/ 1024 + 1;

  vec_kernel<<<blocknums, 1024>>>(nnz, vector);

}





//build grandparents - needs to happen as independent kernel call
uint64_t * build_grandparents(uint64_t nnz, uint64_t * parents){

  uint64_t * grandparents;

  hipMalloc((void **)&grandparents,nnz*sizeof(uint64_t));

  uint64_t blocknums = (nnz -1)/1024 + 1;

  setGrandparents<<<blocknums,1024>>>(nnz, parents, grandparents);

  return grandparents;


}



__global__ void parent_star_gp_compare(uint64_t nnz, uint64_t*parents, uint64_t* grandparents, bool* stars){

  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;

  uint64_t gp = grandparents[tid];
  uint64_t parent = parents[tid];

  if (gp == parent){
    stars[tid] = false;
  }


}

__global__ void star_parent(uint64_t nnz, uint64_t*parents, bool* stars){

  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;

  uint64_t parent = parents[tid];

  stars[tid] = stars[parent];


}



//update stars based on AS starcheck
//simpler version
void parent_star_check(uint64_t nnz, uint64_t * parents, bool *stars){

  uint64_t blocknums = (nnz -1)/1024 + 1;

  //first, build grandparents and reset star
  reset_star<<<blocknums, 1024>>>(nnz, stars);
  uint64_t * grandparents = build_grandparents(nnz, parents);

  //next step
  //if gp[v] != p[v]
  //star[v] and star[gp[v]] = false;
  parent_star_gp_compare<<<blocknums, 1024>>>(nnz, parents, grandparents, stars);

  //inherit parent's condition
  hipFree(grandparents);



}


void cc(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, std::vector<uint64_t> outputRows, uint64_t maxOut, char*kmerVals, uint64_t*kmerLens, uint64_t * kmerParents){


  uint64_t blocknums = (nnz -1)/1024 + 1;
  uint64_t block_vert = (num_vert -1)/1024 + 1;


  //for each vertex init with parent
  char * contigs;
  uint64_t * contig_lens;


  hipMallocManaged((void **)&contigs,num_vert*MAX_VEC*sizeof(char));

  hipMallocManaged((void **)&contig_lens,num_vert*sizeof(uint64_t));

  //expose some extra memory so we don't get weird overwrite bugs
  char * contigs_holder;
  uint64_t * contig_lens_holder;

  hipMallocManaged((void **)&contigs_holder,num_vert*MAX_VEC*sizeof(char));

  hipMallocManaged((void **)&contig_lens_holder,num_vert*sizeof(uint64_t));



  //init both
  printf("If failure, Below this.\n");
  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);
  hipDeviceSynchronize();
  printf("Things worked out\n");
  init_contigs<<<blocknums, 1024>>>(nnz,num_vert, Arows, Acols, Avals, contigs, contig_lens);



  uint64_t * parents;

  hipMalloc((void **)&parents,num_vert*sizeof(uint64_t));

  uint64_t * old_parents;

  hipMalloc((void **)&old_parents,num_vert*sizeof(uint64_t));

  init_parent<<<block_vert, 1024>>>(num_vert, parents);

  //copy over to check
  uint_copy_kernel<<<block_vert, 1024>>>(old_parents, parents, num_vert);

  //init stars
  bool * stars;

  hipMalloc((void ** )&stars, num_vert*sizeof(bool));

  reset_star<<<block_vert, 1024>>>(num_vert, stars);


  uint64_t * grandparents;

  uint64_t iters = 0;


  // printf("stars\n");
  // printCudaStars(num_vert, stars);

  //start with conditional hook
  //this encodes the connections between vertices
  naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);

  //print statements - use with device syncronize
  hipDeviceSynchronize();
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }

  //after unconditional hook, we need to add back in the starts so that the final items aren't one kmer short
  update_leads<<<blocknums, 1024>>>(nnz, contigs, contig_lens, maxOut, kmerVals, kmerLens, kmerParents);


  hipDeviceSynchronize();
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }



  //printf("Before\n");
  //printCudaVec(num_vert, parents);

  //parent_star_check(num_vert, parents, stars);
  bool converged= false;


  do  {

    // printf("Before\n");
    // printCudaVec(num_vert, parents);
    // printf("stars\n");
    // printCudaStars(num_vert, stars);

    //main code
    grandparents = build_grandparents(num_vert, parents);
    parent_cond_hook<<<block_vert, 1024>>>(num_vert, parents, grandparents, stars, contigs, contig_lens, contigs_holder, contig_lens_holder);
    //naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);

    //update contigs
    char * temp = contigs;
    uint64_t * temp_lens = contig_lens;
    contigs = contigs_holder;
    contig_lens = contig_lens_holder;

    contigs_holder = temp;
    contig_lens_holder = temp_lens;

    parent_star_check(num_vert, parents, stars);

    //naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);


    printLenskern(outputRows, contig_lens);
    //shortcutting<<<blocknums, 1024>>>(nnz,parents,grandparents, stars);

    //printf("after\n");
    //printCudaVec(num_vert, parents);
    // printf("stars\n");
    // printCudaStars(num_vert, stars);

    //this is buggy
    //appears to not be consistent
    //reduce_parents<<<block_vert, 1024>>>(parents, old_parents, num_vert);

    //copy to uint64_t before deletion.
    //cudaMemcpy(&count, old_parents, sizeof(uint64_t), cudaMemcpyDeviceToHost);
    converged = starConverged(num_vert, stars);

    //and reset parents
    uint_copy_kernel<<<block_vert, 1024>>>(old_parents, parents, num_vert);
    hipFree(grandparents);

    //printf("Counts\n");
    //count_contigs<<<blocknums, 1024>>>(nnz, parents);
    //printf("Done with iteration %llu: %llu %llu \n", iters, count, num_vert>>(iters-3));
    printf("Done with iter %llu\n", iters);
    iters++;

  } while (!converged);

  printf("Converged\n");
  //if we've really converged we need to syncronize so that the lens are guaranteed
  hipDeviceSynchronize();


  //when done, print outrows
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }

  //time to write to output
  std::ofstream fout;
  fout.open("output.txt");

  for (int i = 0; i < outputRows.size(); i++){

    uint64_t row = outputRows.at(i);
    cout << "len: " << contig_lens[row];

    if (contig_lens[row] >= MAX_VEC){
      cout << " TOO LARGE";
    }
    cout <<  endl;
    for (uint64_t j = 0; j < contig_lens[row]; j++){
      fout << contigs[row*MAX_VEC+j];
    }
    fout << endl;

  }
  fout.close();

  //parents are converged
  hipFree(old_parents);


}
