#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <string>
#include <inttypes.h>
#include <iostream>
#include <vector>
#include <assert.h>
#include <fstream>
#include <chrono>
//Init cuda here
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

//includes needed for 
#include <map>
#include "read_kmers.hpp"
#include "kmer_t.hpp"
#include "cudaCounter.hpp"
#include "noLock_cudaHashMap.hpp"

#ifndef MAX_VEC
#define MAX_VEC 8000
#endif

using namespace std;

//struct for transform
struct kmer_to_pkmer : public thrust::unary_function<kmer_pair,pkmer_t>
{
  __host__ 
  pkmer_t operator()(kmer_pair x) { return x.next_kmer(); }
};

struct kmer_to_start : public thrust::unary_function<kmer_pair,pkmer_t>
{
  __host__ 
  pkmer_t operator()(kmer_pair x) { return x.last_kmer(); }
};



#ifndef CUDA_CHECK
#define CUDA_CHECK(ans)                                                                  \
        gpuAssert((ans), __FILE__, __LINE__);
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
}
}
#endif

typedef unsigned long long int uint64_cu;


//copied host code for handling kmers


//building off of 'perf' adj matrix
//this constructs the matrix from first available
//TODO: parallelize: this should be embarassingly parallel with just a tad of locking
__host__ std::vector<std::pair<kmer_pair, uint64_t>> build_adj_mat(std::vector<kmer_pair> kmers, uint64_t * nnz, char ** vals, uint64_t ** rows, uint64_t ** cols){




  //main kmer counter - at the end of all of this it should equal kmers.size;
  uint64_t counter = 0;

  //the map we'll need: kmer->uint64_t
  std::map<std::string, uint64_t> kmer_to_num;

  uint64_t _nnz = 0;

  //start by adding all kmers to map from pkmer_t - kmer_pair
  //no this is not very efficient O(log n) vs O(1), but this is for debugging only
  //The original issue I suspect stems from something funky with a hash of a hash,
  //so I would like to avoid that as much as possible.
  for (uint64_t i = 0; i < kmers.size(); i++){
    //printf("current: %s\n", kmers[i].kmer.get().c_str());
    //can this be extended into the main loop?
    if (kmers[i].forwardExt() != 'F'){
      //valid edge!
      _nnz +=1;
    }
  }

  printf("Forward pass done with nnz %llu\n", _nnz);

  std::vector<std::pair<kmer_pair, uint64_t>> starts;

  //pass back - we want this for the next step
  *nnz = _nnz;

  //now allocate mat
  char * _vals = new char[_nnz];
  uint64_t * _rows = new uint64_t[_nnz];
  uint64_t * _cols = new uint64_t[_nnz];
  uint64_t slot = 0;

  //iterate through all kmers to find starts
  for (uint64_t i = 0; i < kmers.size(); i++){

    kmer_pair next_kmer = kmers[i];

    //continue on next instruction
    //all instructions valid
    if (next_kmer.forwardExt() == 'F') continue;



    //start file write!


    //start of the main loop
    uint64_t my_val;
    uint64_t next_val;

    //set value of main kmer if DNE
    if (kmer_to_num.count(next_kmer.kmer.get()) == 0){
        kmer_to_num[next_kmer.kmer.get()] = counter;
        counter++;
    }

    //use value stored in map
    my_val = kmer_to_num[next_kmer.kmer.get()];

    //set value of next kmer if DNE
    if (kmer_to_num.count(next_kmer.next_kmer().get()) == 0){
        kmer_to_num[next_kmer.next_kmer().get()] = counter;
        counter++;
    }
    //spooky bug part not needed, not spooky

    next_val = kmer_to_num[next_kmer.next_kmer().get()];

    //mats are allocated, insert into next available slot
    assert (slot < _nnz);

    _vals[slot] = next_kmer.forwardExt();
    _rows[slot] = my_val;
    _cols[slot] = next_val;

    //check if start
    if (next_kmer.backwardExt() == 'F'){
      //save to starts
      //this info is a tad redundant, maybe reduce to just string
      starts.push_back(std::make_pair(next_kmer, slot));
    }

    slot++;

    //update kmer to the next one in the chain
    //next_kmer = kmer_to_kmer[next_kmer.next_kmer().get()];





  }

  printf("Regular construct done, counter is %llu\n", counter);
  fflush(stdout);


  //close sample solution

  //set output
  *vals = _vals;
  *rows = _rows;
  *cols = _cols;

  return starts;

}

//take the starts from a perf construction and insert them into cuda mats
__host__ void prep_starts(std::vector<std::pair<kmer_pair, uint64_t>> starts, uint64_t * rows, uint64_t * startsNnz, char ** startVals, uint64_t** startLens, uint64_t ** startRows){


  uint64_t _startsNnz = starts.size();

  char * _startVals;
  uint64_t * _startLens;
  uint64_t * _startRows;

  //hipMallocManaged for debugging, replace with hipMemcpy for speed later
  hipMallocManaged((void **)&_startVals, _startsNnz*MAX_VEC*sizeof(char));

  hipMallocManaged((void **)&_startLens, _startsNnz*sizeof(uint64_t));

  hipMallocManaged((void **)&_startRows, _startsNnz*sizeof(uint64_t));


  //iterate through outKmers

  for(int i=0; i < starts.size(); i++){

    //can never be too safe
    assert (i < _startsNnz);

    uint64_t slot = std::get<1>(starts.at(i));
    kmer_pair contig_start = std::get<0>(starts.at(i));

    pkmer_t kmer = contig_start.kmer;
    for(int j=0; j < kmer.get().size(); j++){

      //index into the results
      _startVals[i*MAX_VEC+j] = kmer.get()[j];



    }
    _startLens[i] = kmer.get().size();

    //set parent via cond hook
    _startRows[i] = rows[slot];


  }


  //done iterating, output to vecs
  *startsNnz = _startsNnz;
  *startVals = _startVals;
  *startLens = _startLens;
  *startRows = _startRows;


}

__host__ void copy_to_cuda(uint64_t nnz, char * originalVals, uint64_t * originalRows, uint64_t* originalCols, char** newVals, uint64_t ** newRows, uint64_t ** newCols){

  //malloc space for cuda Arrays
  char * _newVals;
  uint64_t * _newRows;
  uint64_t * _newCols;

  hipMalloc((void ** )&_newVals, nnz*sizeof(char));
  hipMalloc((void ** )&_newRows, nnz*sizeof(uint64_t));
  hipMalloc((void ** )&_newCols, nnz*sizeof(uint64_t));



  //memcopys
  hipMemcpy(_newVals, originalVals, nnz*sizeof(char), hipMemcpyHostToDevice);

  hipMemcpy(_newRows, originalRows, nnz*sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(_newCols, originalCols, nnz*sizeof(uint64_t), hipMemcpyHostToDevice);

  *newVals = _newVals;
  *newRows = _newRows;
  *newCols = _newCols;
  //throw in a syncronize just in case, this one is probably clear to remove but you never know
  hipDeviceSynchronize();
}

//generate a list of the vector ids to point to for output
__host__ std::vector<uint64_t> gen_outRows(std::vector<std::pair<kmer_pair, uint64_t>> starts, uint64_t * rows){

  std::vector<uint64_t> outRows;

  for (int i = 0; i < starts.size(); i++){

    uint64_t slot = std::get<1>(starts.at(i));

    outRows.push_back(rows[slot]);
  }

  return outRows;


}


//have everyone attempt to insert into the hashmap
__global__ void insert_all_onethread(uint64_t nnz, kmer_pair* kmers, cudaHashMap * map){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;


  for (uint64_t i =0; i < nnz; i++){

    kmer_pair my_kmer = kmers[i];

    map->insert(my_kmer.kmer, i);

  }
  


}

__global__ void insert_all(uint64_t nnz, kmer_pair* kmers, cudaHashMap * map){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  kmer_pair my_kmer = kmers[tid];

  map->insert(my_kmer.kmer, tid);



}

__global__ void assertInserts(uint64_t nnz, kmer_pair* kmers, cudaHashMap * map){


  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  kmer_pair my_kmer = kmers[tid];

  
  uint64_t val = map->get(my_kmer.kmer);

  if (val == map->size+1){

    printf("Kmer %llu failed to retreive, had val: %llu\n", tid, val);

    uint64_t val2 = map->get(my_kmer.kmer);

    printf("Kmer %llu val 2: %llu\n", tid, val2);


  }


}

__global__ void assertInserts_onethread(uint64_t nnz, kmer_pair* kmers, cudaHashMap * map){


  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid !=0) return;

  for (uint64_t i = 0; i < nnz; i++){


  kmer_pair my_kmer = kmers[i];

  
  uint64_t val = map->get(my_kmer.kmer);

  if (val == map->size+1){

    printf("Kmer %llu failed to retreive, had val: %llu\n", i, val);

    uint64_t val2 = map->get(my_kmer.kmer);

    printf("Kmer %llu val 2: %llu", i, val2);


  }

}


}

//end of copied code for handlers

__device__ uint16_t get_lock_nowait(uint32_t * locks, int index) {
  //set lock to 1 to claim
  //returns 0 if success
  uint32_t zero = 0;
  uint32_t one = 1;
  return atomicCAS(&locks[index], zero, one);
}

__device__ void get_lock(uint32_t * locks, int index) { 
  
  uint16_t result = 1;

  do {
    result = get_lock_nowait(locks, index);
  } while (result !=0);

}

__device__ void free_lock(uint32_t * locks, int index) {

  //set lock to 0 to release
  uint32_t zero = 0;
  uint32_t one = 1;
  //TODO: might need a __threadfence();
  atomicCAS(&locks[index], one, zero);

}

__global__ void counterComp(cudaCounter * counter, uint64_t nnz, uint64_t * counter_holder, uint32_t * locks){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  uint64_t my_val = counter->get(tid);

  if (my_val > nnz) return;


  assert(my_val < nnz+1000);

  while (true){

    uint16_t result = get_lock_nowait(locks, my_val);

    if (result ==0){

      if (counter_holder[my_val] == 0){

        counter_holder[my_val] = 1;

      } else {

        printf("Thread %llu with val %llu received a counter to a filled index\n", tid, my_val);

      }

    free_lock(locks, my_val);

    return;

    }

  }

  

}

__global__ void counterCheck(cudaCounter * counter){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  uint64_t my_val = counter->get(tid);

  //printf("%llu got val %llu\n", tid, my_val);

}

__global__ void counterAtomicCheck(uint64_t * counter){


  //uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;


  atomicAdd((uint64_cu *) counter, 1);

}

//test cases for counter

//how bad is a regular atomic?
__host__ void testAtomic(){

  //uint64_t nnz = 100000;

  uint64_t * counter;

  hipMalloc((void **)&counter, sizeof(uint64_t));

  counterAtomicCheck<<<10000000,100>>>(counter);
}

__host__ void testCounter(){

  uint64_t nnz = 100000;

  cudaCounter * counter;

  initCounter(&counter);

  uint64_t * counter_holder;
  uint32_t * locks;

  CUDA_CHECK(hipMalloc((void**)&counter_holder, (nnz+1000)*sizeof(uint64_t)));
  CUDA_CHECK(hipMalloc((void**)&locks, (nnz+1000)*sizeof(uint32_t)));

  CUDA_CHECK(hipMemset(counter_holder, 0, (nnz+1000)*sizeof(uint64_t)));
  CUDA_CHECK(hipMemset(locks, 0, (nnz+1000)*sizeof(uint32_t)));

  counterComp<<<nnz, 10>>>(counter, nnz, counter_holder, locks);
  hipDeviceSynchronize();

  CUDA_CHECK(hipFree(counter_holder));
  CUDA_CHECK(hipFree(locks));
  freeCounter(counter);

}

//ask for some numbers from the counter
__host__ void testCounterNoCheck(){


  cudaCounter * counter;
  initCounter(&counter);

  counterCheck<<<10000000,100>>>(counter);

  freeCounter(counter);

}



void printrowkern(uint64_t row, char * vec, uint64_t*lengths){

    std::cout << "len: " << lengths[row] << endl;
    std::cout << "[ ";
    for (int j = 0; j < lengths[row]; j++){
      cout << vec[row*MAX_VEC+j];
    }
    std:: cout << " ]" << endl;

}

void printLenskern(std::vector<uint64_t> rows, uint64_t*lengths){

  std::cout << "[ ";
    for (int j = 0; j < rows.size(); j++){
      std::cout << rows.at(j) << ": " << lengths[rows.at(j)] << ", ";
    }
    std:: cout << " ]" << endl;

}

void printCudaVec(uint64_t nnz, uint64_t* cudaVec){

  uint64_t * copy;

  copy = new uint64_t[nnz];

  hipMemcpy(copy, cudaVec,  nnz*sizeof(uint64_t), hipMemcpyDeviceToHost);

  for (uint64_t i =0; i < nnz; i++){
    cout << i << ": [ " << copy[i] << " ]  ";

    if (i % 5 == 4){
      cout << endl;
    }
  }
  cout << endl;

  delete copy;
}

void printCudaStars(uint64_t nnz, bool* cudaVec){

  bool * copy;

  copy = new bool[nnz];

  hipMemcpy(copy, cudaVec,  nnz*sizeof(bool), hipMemcpyDeviceToHost);

  for (uint64_t i =0; i < nnz; i++){
    cout << i << ": [ " << copy[i] << " ]" << endl;
  }

  delete copy;
}

//convert sparse char mat to boolean ints
__global__ void mat_char_to_int(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char* Avals, uint64_t *Bcols, uint64_t * Brows, uint64_t * Bvals){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  Brows[tid] = Arows[tid];
  Bcols[tid] = Acols[tid];
  Bvals[tid] = 1;
}


//initialize every thread to be it's own parent
__global__ void init_parent(uint64_t nnz,  uint64_t* parent){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    parent[tid] = tid;
  }

}

//the counter variable is significantly less than it should be - why?
__global__ void set_parents_from_hashmap(uint64_t nnz, kmer_pair* kmers, pkmer_t * pkmers, uint64_t * parents, char * extensions, cudaHashMap * map, uint64_t * count){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  //query myself and my parents
  uint64_t my_slot = map->get(kmers[tid].kmer);

  //this would be a bug
  assert(my_slot != map->size+1);

  //this must fail at least once - otherwise how do you know the kmer ended?
  uint64_t my_parent = map->get(pkmers[tid]);
  //uint64_t my_parent = map->get(kmers[tid].next_kmer());

  if(my_parent != map->size+1){



    parents[my_slot] = my_parent;

    extensions[my_slot] = kmers[tid].forwardExt();

    if(extensions[my_slot] == 'F'){

      printf("Tid failed: kmer %llu with forwardExt %c claims to have parent at %llu\n", my_slot, kmers[tid].forwardExt(), my_parent);
      map->get(pkmers[tid]);

    }

   

    return;

  } else {

    assert(kmers[tid].forwardExt() == 'F');
    atomicAdd((long long unsigned int *) count, (long long unsigned int ) 1);

  }


}

__global__ void print_count(uint64_t * count){


  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid == 0) printf("Num starts: %llu\n", *count);

}

__host__ uint64_t prep_parents(uint64_t nnz, kmer_pair* kmers, pkmer_t * pkmers, uint64_t * parents, char * extensions, cudaHashMap * map){


uint64_t blocksize = 1024;
uint64_t num_blocks = (nnz-1)/blocksize+1;

uint64_t * count;
//this is slow but pretty convenient
hipMallocManaged((void **)&count, sizeof(uint64_t));

count[0] = 0;

//initialize every thread to be it's own parent
init_parent<<<num_blocks, blocksize>>>(nnz, parents);


set_parents_from_hashmap<<<num_blocks, blocksize>>>(nnz, kmers, pkmers, parents, extensions, map, count);

hipDeviceSynchronize();

print_count<<<1,1>>>(count);

hipDeviceSynchronize();

uint64_t to_return = *count;

hipFree(count);

return to_return;

}


__global__ void find_starts_kernel(uint64_t nnz, pkmer_t * pkmers, uint64_t * starts, uint64_t * counter, cudaHashMap* hashMap){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  if (hashMap->get(pkmers[tid]) == hashMap->size+1){

    uint64_t my_index = atomicAdd((unsigned long long int  *) counter, (long long unsigned int) 1);

    starts[my_index] = tid;

  }


}

__host__ void find_starts_cuda(uint64_t nnz, pkmer_t * pkmers, uint64_t startNnz, uint64_t ** startIds, cudaHashMap * map){


  uint64_t * counter;
  uint64_t * starts;

  hipMallocManaged((void **)&counter, sizeof(uint64_t));
  hipMallocManaged((void **)&starts, sizeof(uint64_t)*startNnz);


  counter[0] = 0;
  uint64_t blocksize = 1024;
  uint64_t num_blocks = (nnz-1)/blocksize+1;

  find_starts_kernel<<<num_blocks, blocksize>>>(nnz, pkmers, starts, counter, map);


  print_count<<<1,1>>>(counter);

  hipFree(counter);

  *startIds = starts;

}

//discern parents from pkmers


// __global__ void naive_cond_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char * Avals, uint64_t * parent, bool * star){

//   uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

//   if (tid >= nnz) return;

//   uint64_t u = Arows[tid];
//   uint64_t v = Acols[tid];

//   uint64_t parent_u = parent[u];
//   uint64_t parent_v = parent[v];


//   //retreive f earlier
//   uint64_t gparent_u = parent[parent[u]];
//   uint64_t old;

//   //star hook procedure
//   if (star[u] && parent[u] > parent[v]){
//     old = (uint64_t) atomicCAS( (uint64_cu *) parent+parent_u, (uint64_cu) gparent_u, (uint64_cu) parent_v);
//     //if this is the case we must have succeeded
//     if (old == gparent_u){
//       return;
//     }
//     parent_v = parent[v];
//     parent_u = parent[u];
//     gparent_u = parent[parent_u];
//   }


// }

__global__ void parent_cond_hook_no_branch(uint64_t nnz, uint64_t * parent, uint64_t * parent_holder, uint64_t * gparent, bool * star, char* contigs, uint64_t * contig_lens, char* contigs_holder, uint64_t * contig_lens_holder){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  //for parent cond hook, if I am not a star, set my parent to my grandparent

  //if star[u]
  // parent[u] = parent[parent[u]]
  uint64_t gparent_u = gparent[tid];
  uint64_t parent_u = parent[tid];

  if (star[tid]){

    //absorb from your parent
    //first copy over your material

    //compress parents
    uint64_t my_contig_len = contig_lens[tid];
    uint64_t my_parent_len = contig_lens[parent_u];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_parent = contigs+MAX_VEC*parent_u;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    //copy from my parent
    for (int i =0; i < my_parent_len; i++){
      my_output[i+my_contig_len] = my_parent[i];
    }

    //copy to new len
    contig_lens_holder[tid] = my_contig_len+my_parent_len;

    //and absorb
    parent_holder[tid] = gparent_u;


    //having a branch here is really bad and should  not happen
  }


}

//following
__global__ void map_contigs(uint64_t maxOut, char * startVals, uint64_t * startLens, uint64_t * startParents, uint64_t * contig_index, uint64_t * parents, uint64_t * contig_map, uint64_t * contig_map_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= maxOut) return;

  //now find correct length and copy over into buffer
  uint64_t my_parent = startParents[tid];
  uint64_t my_contig = parents[my_parent];

  contig_map[tid] = my_contig;
  contig_map_lens[tid] = contig_index[my_parent] + startLens[tid];


}


//output the mappings
__global__ void print_mappings(uint64_t maxOut, uint64_t * contig_map, uint64_t * contig_map_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;

  for (uint64_t i =0; i < maxOut; i++){

    printf("Contig %llu -> %llu, len %llu\n", i, contig_map[i], contig_map_lens[i]);
  }

}

__host__ void printBuftype(std::string bufname, void* buffer){

  hipPointerAttribute_t bufStats;
  CUDA_CHECK(hipPointerGetAttributes(&bufStats, buffer));

  cout << "buffer " << bufname << " of type ";

  if (bufStats.type == cudaMemoryTypeUnregistered) cout << "unregistered";
  else if (bufStats.type == hipMemoryTypeHost) cout << "Host";
  else if (bufStats.type == hipMemoryTypeDevice) cout << "Device";
  else if (bufStats.type == hipMemoryTypeManaged) cout << "Managed";
  else cout << "failure on type somehow";

  cout << endl;

}

//this is bad - google cuda heap allocation vs hipMalloc to learn more
//basically this creates memory that is unusable by host
__global__ void mallocContigs(uint64_t maxOut, char ** final_contigs, uint64_t * contig_map_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= maxOut) return;

  char * temp_contig;

  hipMalloc((void **)&temp_contig,contig_map_lens[tid]*sizeof(char));
  //allocate device_side memory


  final_contigs[tid] = temp_contig;


}


//this should work better - create host version and malloc over
__host__ void mallocHostContigs(uint64_t maxOut, char ** final_contigs, uint64_t * contig_map_lens){

  char ** host_final_contigs;
  hipHostMalloc((void **)&host_final_contigs,maxOut*sizeof(char * ));

  uint64_t * _lens; // = (uint64_t *) malloc(maxOut*sizeof(uint64_t));
  CUDA_CHECK(hipHostMalloc((void **)&_lens,maxOut*sizeof(uint64_t)));

  CUDA_CHECK(hipMemcpy(_lens, contig_map_lens, maxOut*sizeof(uint64_t), hipMemcpyDeviceToHost));


  for (uint64_t tid =0; tid < maxOut; tid++){


    char * temp_contig;

    CUDA_CHECK(hipMalloc((void **)&temp_contig,_lens[tid]*sizeof(char)));
    CUDA_CHECK(hipMemset(temp_contig, 'F', _lens[tid]*sizeof(char)));
    //allocate device_side memory


    host_final_contigs[tid] = temp_contig;

  }

  CUDA_CHECK(hipMemcpy(final_contigs, host_final_contigs, maxOut*sizeof(char *), hipMemcpyHostToDevice));


  CUDA_CHECK(hipHostFree(host_final_contigs));
}


__host__ void freeHostContigs (uint64_t maxOut, char ** final_contigs){

  char ** host_final_contigs;
  hipHostMalloc((void **)&host_final_contigs,maxOut*sizeof(char * ));



  CUDA_CHECK(hipMemcpy(host_final_contigs, final_contigs, maxOut*sizeof(char*), hipMemcpyDeviceToHost));

  for (uint64_t i =0; i < maxOut; i++){
    CUDA_CHECK(hipFree(host_final_contigs[i]));
  }

  hipDeviceSynchronize();

  CUDA_CHECK(hipHostFree(host_final_contigs));

}

__global__ void freeContigs(uint64_t maxOut, char ** final_contigs){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= maxOut) return;

  hipFree(final_contigs[tid]);


}

__global__ void fill_contigs_starts(uint64_t maxOut, char ** final_contigs, char * startVals, uint64_t* startLens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= maxOut) return;

  for (uint64_t i=0; i < startLens[tid]; i++){

    final_contigs[tid][i] = startVals[tid*MAX_VEC+i];


  }

}


//fill all contigs
//each vertex finds its contig
// then its insert position into that contig
// and then writes in parallel
//idea! use 2d blocking
//x is vertex num -- if this is not fast enough do vertex tidy = block.x to have all threads access same value
//y is maxOut num - try all array indices in parallel, kill all threads that don't continue
__global__ void fill_contigs(uint64_t num_verts, char ** final_contigs, uint64_t maxOut, uint64_t * contig_map, uint64_t * contig_map_lens, char * contigs, uint64_t * contig_index, uint64_t * contig_lens, uint64_t * parents){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  uint64_t tidy = threadIdx.y + blockIdx.y*blockDim.y;

  //threads are launching, just not inputting correctly


  if (tid >= num_verts) return;
  if (tidy >= maxOut) return;


  uint64_t my_contig = parents[tid];

  uint64_t my_index_contig = contig_map[tidy];

  //printf("Thread %llu, %llu launching, comp %llu, %llu with len %llu\n", tid, tidy, my_contig, my_index_contig, contig_lens[tid]);


  if (my_contig != my_index_contig) return;

  if (contig_lens[tid] == 0){
    return;
  }



  //printf("Vertex %llu lines up with contig %llu\n", tid, tidy);

  //tidy is now the contig to access
  uint64_t my_start = contig_map_lens[tidy] - contig_index[tid];

  //correct index is 593
  final_contigs[tidy][my_start] = contigs[tid];

}

__global__ void move_contigs_to_host(uint64_t maxOut, char** final_contigs, char** host_final_contigs, uint64_t * contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= maxOut) return;

  //find my id, and memcpy async
  hipMemcpyAsync(host_final_contigs[tid], final_contigs[tid], contig_lens[tid]*sizeof(char), hipMemcpyDeviceToHost);


}

__global__ void check_host_contig(char * contig, uint64_t len){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;

  printf("Dev contig from host pointer:\n");
  for (uint64_t i = 0; i < len; i++){
    printf("%c", contig[i]);
  }
  printf("\n");

}

//copy over contigs to host, then save to a file
__host__ void save_contigs(std::string filename, uint64_t maxOut, char ** final_contigs, uint64_t * final_lens){

  //the first step is to copy over the lengths, so we know how large of a buffer to allocate

  uint64_t * _lens; // = (uint64_t *) malloc(maxOut*sizeof(uint64_t));
  CUDA_CHECK(hipHostMalloc((void **)&_lens,maxOut*sizeof(uint64_t)));

  char ** host_final_contigs; // = (char **) malloc(maxOut*sizeof(char *));
  CUDA_CHECK(hipHostMalloc((void **)&host_final_contigs,maxOut*sizeof(char * )));

  CUDA_CHECK(hipMemcpy(_lens, final_lens, maxOut*sizeof(uint64_t), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipMemcpy(host_final_contigs, final_contigs, maxOut*sizeof(char * ), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  std::ofstream fout;
  fout.open(filename);

  // printBuftype("lens", _lens);
  // printBuftype("host_final_contigs", host_final_contigs);

  for (uint64_t i =0 ; i < maxOut; i++){

    char * buffer; //= (char * ) malloc(_lens[i]*sizeof(char));
    CUDA_CHECK(hipHostMalloc((void **)&buffer,_lens[i]*sizeof(char)));

    //buffer = host_final_contigs[i];

    // printBuftype("temp_buf", buffer);


    char * cudaBuf = host_final_contigs[i];

    // printBuftype("host_final_contigs[i]", host_final_contigs[i]);
    // printBuftype("cudaBuf", cudaBuf);

    //printf("Copy size vs copy: %llu vs %llu \n", _lens[i], _lens[i]*sizeof(char));

    CUDA_CHECK(hipMemcpy(buffer, cudaBuf, _lens[i]*sizeof(char), hipMemcpyDefault));


    //hipDeviceSynchronize();
    //host final contig is still a cuda pointer
    //so have a thread read the device memory
    //check_host_contig<<<1,1>>>(host_final_contigs[i], _lens[i]);

    //hipDeviceSynchronize();
    fflush(stdout);


    //file write
    //cout << "Contig:" << i << " with len " << _lens[i] <<  endl;
    for (uint64_t j=0; j < _lens[i]; j++){
      //cout << buffer[i];
      //printf("%c", buffer[j]);
      fout << buffer[j];
    }
    //printf("\n");
    fout << endl;

    fflush(stdout);
    //done with save
    CUDA_CHECK(hipHostFree(buffer));
  }

  CUDA_CHECK(hipHostFree(_lens));
  CUDA_CHECK(hipHostFree(host_final_contigs));

}

__global__ void check_contig(uint64_t contig_id, char ** final_contigs, uint64_t * lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;

  //printf("Looking at contig %llu with len %llu\n", contig_id, lens[contig_id]);

  for (uint64_t i=0; i <lens[contig_id]; i++){
    printf("%c", final_contigs[contig_id][i]);
  }
  printf("\n");
}

//fill the start of every contig based input starts
__global__ void parent_cond_hook(uint64_t nnz, uint64_t * parent, uint64_t * parent_holder, uint64_t * gparent, bool * star, char* contigs, uint64_t * contig_lens, char* contigs_holder, uint64_t * contig_lens_holder){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  //for parent cond hook, if I am not a star, set my parent to my grandparent

  //if star[u]
  // parent[u] = parent[parent[u]]
  uint64_t gparent_u = gparent[tid];
  uint64_t parent_u = parent[tid];

  if (star[tid]){

    //absorb from your parent
    //first copy over your material

    //compress parents
    uint64_t my_contig_len = contig_lens[tid];
    uint64_t my_parent_len = contig_lens[parent_u];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_parent = contigs+MAX_VEC*parent_u;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    //copy from my parent
    for (int i =0; i < my_parent_len; i++){
      my_output[i+my_contig_len] = my_parent[i];
    }

    //copy to new len
    contig_lens_holder[tid] = my_contig_len+my_parent_len;

    //and absorb
    parent_holder[tid] = gparent_u;


    //having a branch here is really bad and should  not happen
  } else {


    //TODO: Move this section to the star check
    //atm we are repeating work

    uint64_t my_contig_len = contig_lens[tid];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    contig_lens_holder[tid] = my_contig_len;

    parent_holder[tid] =  parent_u;

  }
  // if (parent[tid] =  parent[parent[tid]]){
  //   star[tid] = false;
  // }


}

__global__ void len_cond_hook(uint64_t nnz, uint64_t * parent, uint64_t * parent_holder, bool * star, uint64_t * contig_index, uint64_t * contig_index_holder){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  //for parent cond hook, if I am not a star, set my parent to my grandparent

  //if star[u]
  // parent[u] = parent[parent[u]]

  uint64_t parent_u = parent[tid];
  uint64_t gparent_u = parent[parent_u];
  uint64_t parent_len = 0;
  uint64_t my_contig_index = contig_index[tid];

  if (star[tid]){

    //absorb from your parent
    //first copy over your material

    //compress parents

    parent_len = contig_index[parent_u];

  }


    //copy to new len
    contig_index_holder[tid] = my_contig_index+parent_len;

    //and absorb
    parent_holder[tid] = gparent_u;

}

//perform initial setup
//given an adj matrix,
__global__ void simple_adj_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char* Avals, uint64_t * parent, bool * stars, char* contigs, uint64_t * contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  uint64_t my_index = Arows[tid];

  parent[my_index] = Acols[tid];

  //and init contigs
  char my_val = Avals[tid];
  contigs[my_index*MAX_VEC] = my_val;
  contig_lens[my_index] = 1;

}

__global__ void len_adj_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char* Avals, uint64_t * parent, bool * stars, char* contigs, uint64_t * contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= nnz) return;

  uint64_t my_index = Arows[tid];

  parent[my_index] = Acols[tid];

  //and init contigs
  char my_val = Avals[tid];
  contigs[my_index] = my_val;
  contig_lens[my_index] = 1;

}

//the sum of contig sections should be constant
__global__ void count_bases(uint64_t nnz, char * contigs){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;

  uint64_t counter = 0;
  for (uint64_t i =0; i < nnz; i++){
    if (contigs[i] == 'A' || contigs[i] == 'C' || contigs[i] == 'T' || contigs[i] == 'G'){
      counter +=1;
    }
  }
  printf("Conting counter: %llu\n", counter);

}

__global__ void sum_lens(uint64_t nnz, uint64_t * lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid != 0) return;

  uint64_t counter = 0;
  for (uint64_t i =0; i < nnz; i++){
    counter += lens[i];
  }
  printf("lens counter: %llu\n", counter);

}

//unconditional hook - this is frankly bizarre
//and im not sure how it's 'worked' so far
// __global__ void naive_uncond_hook(uint64_t nnz, uint64_t * Arows, uint64_t * Acols, char * Avals, uint64_t * parent, bool * star){

//   uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

//   if (tid >= nnz) return;

//   uint64_t u = Arows[tid];
//   uint64_t v = Acols[tid];

//   uint64_t parent_u = parent[u];
//   uint64_t parent_v = parent[v];

//   //retreive f earlier
//   uint64_t gparent_u = parent[parent[u]];
//   uint64_t old;

//   //star hook procedure
//   if (star[u] && parent[u] != parent[v]){
//     old = (uint64_t) atomicCAS( (uint64_cu *) parent+parent_u, (uint64_cu) gparent_u, (uint64_cu) parent_v);
//     //if this is the case we must have succeeded
//     if (old == gparent_u){
//       return;
//     }
//     parent_v = parent[v];
//     parent_u = parent[u];
//     gparent_u = parent[parent_u];
//   }


// }

__global__ void shortcutting(uint64_t nnz, uint64_t * parents, uint64_t * gparents, bool * stars){

  //assume gparents already defined
  uint64_t tid = threadIdx.x +  blockIdx.x * blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  uint64_t v = tid;

  //star hook procedure
  if (!stars[v]){

    parents[v] = gparents[v];

  }


}

__global__ void setGrandparents(uint64_t nnz, uint64_t * parents, uint64_t * grandparents){

  //assume gparents already defined
  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  grandparents[tid] = parents[parents[tid]];

  return;

}

__global__ void reset_star(uint64_t nnz, bool * stars){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  //double check this is numcols
  if (tid >= nnz) return;

  stars[tid] = true;


}




//initialize lengths to be 0
__global__ void init_contig_lens(uint64_t nnz,  uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    contig_lens[tid] = 0;
  }

}

//assert lengths are null
__global__ void assert_contig_lens(uint64_t nnz,  uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){
    if (contig_lens[tid] != 0){
      printf("contiig length %llu not 0\n", tid);
    }
  }

}

//initialize lengths to be 0
__global__ void init_contigs(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, char* contigs, uint64_t* contig_lens){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid < nnz){

    //grab row
    char my_val = Avals[tid];
    contigs[Arows[tid]*MAX_VEC] = my_val;
    contig_lens[Arows[tid]] = 1;
  }

}



// __device__ char semiring_multiply(char a, char b){
//
//   printf("Multiplying %c, %c\n", a,b);
//   if (a == 'z' ||  b == 'z')
//     return 'z';
//
//   if (a == 0x20) return b;
//
//   return a;
// }

// __device__ char semiring_add(char a, char b){
//
//   printf("adding %c, %c\n", a,b);
//   if (a == 0x20){
//     return b;
//   }
//   if (b == 0x20){
//     return a;
//   }
//   //both nonzero, bad path
//   //this will corrupt any future adds to this index as well
//   return 'z';
// }

__global__ void copy_kernel(double * to_copy, double* items, size_t n) {
  int tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    to_copy[tid] = items[tid];
  }
}

__global__ void uint_copy_kernel(uint64_t* to_fill, uint64_t* to_copy, size_t n) {
  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    to_fill[tid] = to_copy[tid];
  }
}

__global__ void copy_kernel_char(char * to_copy, char* items, size_t n) {
  int tid = threadIdx.x +  blockIdx.x*blockDim.x;



  if (tid < n) {
    printf("Tid %d reporting\n", tid);
    to_copy[tid] = items[tid];
  }
}

//__global__ void kmer_copy_kernel(uint64_t contig_num, char * contigs, )

//After the conditional hooking step, we should push any updated reads into the contigs
//because this happens first, the len of the contigs must be 1
__global__ void update_leads(uint64_t nnz, char * contigs, uint64_t * contig_lens, uint64_t num_updates, char * updates, uint64_t * update_lens, uint64_t * parent){

  uint64_t tid = threadIdx.x +  blockIdx.x*blockDim.x;

  if (tid >= num_updates) return;


  uint64_t contig_index = parent[tid];

  assert(contig_lens[contig_index] == 1);

  contig_lens[contig_index] += update_lens[tid];

  //move the first intem back to the last index in preparation for the copy kernel
  //0th index to

  //was a -1 on the left
  //lets split this into parts
  contigs[MAX_VEC*contig_index+contig_lens[contig_index]-1] = contigs[MAX_VEC*contig_index];


  //copy kernel moved from cc
  for (int i = 0; i < contig_lens[contig_index]-1; i++){

    contigs[MAX_VEC*contig_index+i] = updates[MAX_VEC*tid+i];

  }

  //finished>
  return;

}



__global__ void vec_kernel(int nnz, int* vec){
  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;


  vec[tid] = 1;
}



__global__ void clear_kernel(int nnz, char*vec){
  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;


  vec[tid] =0x20;

}




__global__ void check_stars(uint64_t nnz,  bool * stars, int* converged){

    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= nnz) return;

    if (stars[tid]){
      //printf("We should not converge: %llu\n", tid);

      //swap 1 with 0 if it hasn't happened
      //come back and time this
      converged[0] = 0;
      //cas works, let's test regular convergence
      //atomicCAS(converged,1,0);

    }

    return;

}



// void  fill_wrapper(int nnz, int*vals, int*rows, int*cols){
//
//   int blocknums  = (nnz - 1)/ 1024 + 1;
//
//   fill_matrix<<<blocknums, 1024>>>(nnz, vals,rows,cols);
//
// }

void copy_wrapper(double * to_copy, double* items, size_t n){

  copy_kernel<<<1,n>>>(to_copy, items, n);

}

//check if all items in vec are false: if true, converged
bool starConverged(uint64_t nnz, bool*stars){

  int * converged;

  hipMallocManaged((void **)&converged,1*sizeof(int));

  //set to true initially
  converged[0] = 1;

  uint64_t blocknums = (nnz -1)/1024 + 1;

  check_stars<<<blocknums, 1024>>>(nnz, stars, converged);
  hipDeviceSynchronize();

  bool result = true;

  result = (converged[0] == 1);

  std::cout << "converged: " << result << "." << std::endl;
  hipFree(converged);

  return result;

}

void  fill_vector(int nnz, int*vector){

  int blocknums  = (nnz - 1)/ 1024 + 1;

  vec_kernel<<<blocknums, 1024>>>(nnz, vector);

}





//build grandparents - needs to happen as independent kernel call
uint64_t * build_grandparents(uint64_t nnz, uint64_t * parents){

  uint64_t * grandparents;

  hipMalloc((void **)&grandparents,nnz*sizeof(uint64_t));

  uint64_t blocknums = (nnz -1)/1024 + 1;

  setGrandparents<<<blocknums,1024>>>(nnz, parents, grandparents);

  return grandparents;


}



__global__ void parent_star_gp_compare(uint64_t nnz, uint64_t*parents, bool* stars){

  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;

  uint64_t parent = parents[tid];
  uint64_t gp = parents[parent];

  if (gp == parent){
    stars[tid] = false;
  }


}

__global__ void parent_star_gp_compare_one_update(uint64_t nnz, uint64_t*parents, bool* stars, char*contigs, uint64_t* contig_lens, char* contigs_holder, uint64_t * contig_lens_holder){

  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;
  //only look at stars that have not reset
  if (!stars[tid]) return;

  uint64_t parent = parents[tid];
  uint64_t gp = parents[parent];

  if (gp == parent){
    stars[tid] = false;

    //and update
    uint64_t my_contig_len = contig_lens[tid];
    char * my_contig = contigs + MAX_VEC*tid;
    char * my_output = contigs_holder + MAX_VEC*tid;

    //copy from me
    for (int i = 0; i < my_contig_len; i++){
      my_output[i] = my_contig[i];
    }

    contig_lens_holder[tid] = my_contig_len;

    //doesn't need to be set - gets propogated in a later copy
    //parent_holder[tid] =  parent_u;

  }


}







__global__ void star_parent(uint64_t nnz, uint64_t*parents, bool* stars){

  int tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= nnz) return;

  uint64_t parent = parents[tid];

  stars[tid] = stars[parent];


}



//update stars based on AS starcheck
//simpler version
//this is soo slow
void parent_star_check(uint64_t nnz, uint64_t * parents, bool *stars){

  uint64_t blocknums = (nnz -1)/1024 + 1;

  //first, build grandparents and reset star
  reset_star<<<blocknums, 1024>>>(nnz, stars);
  //uint64_t * grandparents = build_grandparents(nnz, parents);
  hipDeviceSynchronize();
  //printf("Reset Star\n");
  //fflush(stdout);

  //next step
  //if gp[v] != p[v]
  //star[v] and star[gp[v]] = false;
  parent_star_gp_compare<<<blocknums, 1024>>>(nnz, parents, stars);

  hipDeviceSynchronize();
  //printf("Set stars\n");
  //fflush(stdout);

  //inherit parent's condition
  //hipFree(grandparents);



}

void parent_star_check_noreset(uint64_t nnz, uint64_t * parents, bool *stars, char*contig, uint64_t* contig_lens, char* contig_holder, uint64_t * contig_lens_holder){

  uint64_t blocknums = (nnz -1)/1024 + 1;

  //first, build grandparents and reset star


  //next step
  //if gp[v] != p[v]
  //star[v] and star[gp[v]] = false;
  parent_star_gp_compare_one_update<<<blocknums, 1024>>>(nnz, parents, stars, contig, contig_lens, contig_holder, contig_lens_holder);

  //inherit parent's condition
  //hipFree(grandparents);



}


void cc(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, std::vector<uint64_t> outputRows, uint64_t maxOut, char*kmerVals, uint64_t*kmerLens, uint64_t * kmerParents){


  uint64_t blocknums = (nnz -1)/1024 + 1;
  uint64_t block_vert = (num_vert -1)/1024 + 1;


  //for each vertex init with parent
  char * contigs;
  uint64_t * contig_lens;

  auto start = std::chrono::high_resolution_clock::now();


  hipMallocManaged((void **)&contigs,num_vert*MAX_VEC*sizeof(char));


  hipMallocManaged((void **)&contig_lens,num_vert*sizeof(uint64_t));


  //expose some extra memory so we don't get weird overwrite bugs
  char * contigs_holder;
  uint64_t * contig_lens_holder;

  hipMallocManaged((void **)&contigs_holder,num_vert*MAX_VEC*sizeof(char));

  hipMallocManaged((void **)&contig_lens_holder,num_vert*sizeof(uint64_t));



  //init both
  //printf("If failure, Below this.\n");
  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);

  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens_holder);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens_holder);
  hipDeviceSynchronize();

  auto contig_setup = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> diff = contig_setup-start;

  std::cout << "Time required for contig setup: " << diff.count() << " s\n";
  //printf("Things worked out\n");
  //init_contigs<<<blocknums, 1024>>>(nnz,num_vert, Arows, Acols, Avals, contigs, contig_lens);



  uint64_t * parents;

  hipMalloc((void **)&parents,num_vert*sizeof(uint64_t));

  uint64_t * parents_holder;

  hipMalloc((void **)&parents_holder,num_vert*sizeof(uint64_t));

  init_parent<<<block_vert, 1024>>>(num_vert, parents);
  //init_parent<<<block_vert, 1024>>>(num_vert, parents_holder);

  //copy over to check
  uint_copy_kernel<<<block_vert, 1024>>>(parents_holder, parents, num_vert);

  //init stars
  bool * stars;

  hipMalloc((void ** )&stars, num_vert*sizeof(bool));

  reset_star<<<block_vert, 1024>>>(num_vert, stars);


  uint64_t * grandparents;

  uint64_t iters = 0;


  // printf("stars\n");
  // printCudaStars(num_vert, stars);

  //start with conditional hook
  //this encodes the connections between vertices
  //this isn't right :ADF:ASD
  //naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);
  simple_adj_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars, contigs, contig_lens);



  //print statements - use with device syncronize
  hipDeviceSynchronize();
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }

  //after unconditional hook, we need to add back in the starts so that the final items aren't one kmer short
  update_leads<<<blocknums, 1024>>>(nnz, contigs, contig_lens, maxOut, kmerVals, kmerLens, kmerParents);


  hipDeviceSynchronize();
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }

  auto full_setup = std::chrono::high_resolution_clock::now();

  diff = full_setup-contig_setup;

  std::cout << "Time required for final setup: " << diff.count() << " s\n";


  //printf("Before\n");
  //printCudaVec(num_vert, parents);

  //parent_star_check(num_vert, parents, stars);
  bool converged= false;


  do  {

    auto iter_start = std::chrono::high_resolution_clock::now();

    // printf("Before\n");
    // printCudaVec(num_vert, parents);
    // printf("stars\n");
    // printCudaStars(num_vert, stars);

    //main code
    grandparents = build_grandparents(num_vert, parents);
    parent_cond_hook<<<block_vert, 1024>>>(num_vert, parents, parents_holder, grandparents, stars, contigs, contig_lens, contigs_holder, contig_lens_holder);
    //naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);

    //update contigs
    char * temp = contigs;
    uint64_t * temp_lens = contig_lens;
    contigs = contigs_holder;
    contig_lens = contig_lens_holder;

    contigs_holder = temp;
    contig_lens_holder = temp_lens;

    hipDeviceSynchronize();

    auto cond_hook = std::chrono::high_resolution_clock::now();

    diff = cond_hook-iter_start;

    std::cout << "Time required for cond hook: " << diff.count() << " s\n";


    //trade parents
    //doing it this way lets us reuse the memory efficiently
    uint64_t * temp_parents = parents;
    parents =  parents_holder;
    parents_holder  = temp_parents;

    printf("Entering star check\n");
    fflush(stdout);


    parent_star_check(num_vert, parents, stars);

    //parent_star_check_noreset(num_vert, parents, stars, contigs, contig_lens, contigs_holder, contig_lens_holder);

    hipDeviceSynchronize();

    auto star_hook = std::chrono::high_resolution_clock::now();

    diff = star_hook - cond_hook;

    std::cout << "Time required for star check: " << diff.count() << " s\n";

    //naive_uncond_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars);


    //printLenskern(outputRows, contig_lens);
    //shortcutting<<<blocknums, 1024>>>(nnz,parents,grandparents, stars);

    //printf("after\n");
    //printCudaVec(num_vert, parents);
    // printf("stars\n");
    // printCudaStars(num_vert, stars);

    //this is buggy
    //appears to not be consistent
    //reduce_parents<<<block_vert, 1024>>>(parents, parents_holder, num_vert);

    //copy to uint64_t before deletion.
    //hipMemcpy(&count, parents_holder, sizeof(uint64_t), hipMemcpyDeviceToHost);
    converged = starConverged(num_vert, stars);

    //and reset parents
    uint_copy_kernel<<<block_vert, 1024>>>(parents_holder, parents, num_vert);
    hipFree(grandparents);

    hipDeviceSynchronize();

    //printf("Counts\n");
    //count_contigs<<<blocknums, 1024>>>(nnz, parents);
    //printf("Done with iteration %llu: %llu %llu \n", iters, count, num_vert>>(iters-3));
    printf("Done with iter %llu\n", iters);

    auto iter_end = std::chrono::high_resolution_clock::now();

    diff = iter_end-iter_start;

    std::cout << "Time required for whole iter: " << diff.count() << " s" << endl;

    iters++;

  } while (!converged);

  printf("Converged\n");
  //if we've really converged we need to syncronize so that the lens are guaranteed
  hipDeviceSynchronize();


  //when done, print outrows
  // for (int i =0; i < outputRows.size(); i++){
  //   printrowkern(outputRows.at(i), contigs, contig_lens);
  // }

  //time to write to output
  std::ofstream fout;
  fout.open("cc_output.dat");

  for (int i = 0; i < outputRows.size(); i++){

    uint64_t row = outputRows.at(i);
    cout << "len: " << contig_lens[row];

    if (contig_lens[row] >= MAX_VEC){
      cout << " TOO LARGE";
    }
    cout <<  endl;
    for (uint64_t j = 0; j < contig_lens[row]; j++){
      fout << contigs[row*MAX_VEC+j];
    }
    fout << endl;

  }
  fout.close();

  //last call to assert correctness
  hipDeviceSynchronize();

  //parents are converged
  //free up memory
  hipFree(parents);
  hipFree(parents_holder);
  hipFree(contigs);
  hipFree(contig_lens);
  hipFree(stars);
  hipFree(contigs_holder);
  hipFree(contig_lens_holder);


}

//iteratively solve the cc problem by jumping through the parents array
__global__ void cuda_solver_kernel(uint64_t nnz, uint64_t num_vert, char* contigs, uint64_t* contig_lens, uint64_t * parents, uint64_t startNnz, char * startVals, uint64_t * startLens, uint64_t * startRows){

  uint64_t tid = threadIdx.x +  blockIdx.x * blockDim.x;

  if (tid >= startNnz) return;

  uint64_t row = startRows[tid];

  //else
  printf("Tid %llu working on kmer that starts at %llu\n", tid, row);

  //first step, copy over my parent - if we have a typo it may show up here
  assert(contig_lens[startRows[tid]] == 1);

  //now steal character
  char my_first_extension = contigs[row*MAX_VEC];

  //and fill in
  for (uint64_t i = 0; i < startLens[tid]; i++){

    contigs[row*MAX_VEC +i] = startVals[tid*MAX_VEC +i];
  }
  contigs[row*MAX_VEC + startLens[tid]] = my_first_extension;
  contig_lens[row] += startLens[tid];

  //start looks good, lets test the rest!
  uint64_t my_parent = parents[row];
  while (my_parent != parents[my_parent]){

    //with 0 mutations, everyone ahead of me should have exactly 1 base
    assert(contig_lens[my_parent] == 1);
    //copy over data
    for (uint64_t i = 0; i < contig_lens[my_parent]; i++){

      contigs[row*MAX_VEC+contig_lens[row]+i] = contigs[my_parent*MAX_VEC + i];

    }
    //update length
    contig_lens[row] += contig_lens[my_parent];

    //and update parent
    parents[row] = parents[parents[row]];
    my_parent = parents[row];

  }

}


//separate the approach a little from cc
//simplest version does a one off trace - builds all contigs in parallel locally
//this now works!
void iterative_cuda_solver(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, std::vector<uint64_t> outputRows, uint64_t outNnz, char*kmerVals, uint64_t*kmerLens, uint64_t * kmerParents){

  //block sizes for mat and vert based ops
  uint64_t blocknums = (nnz -1)/1024 + 1;
  uint64_t block_vert = (num_vert -1)/1024 + 1;

  //define some memory to work with
  char * contigs;
  uint64_t * contig_lens;

  auto start = std::chrono::high_resolution_clock::now();

  hipMallocManaged((void **)&contigs,num_vert*MAX_VEC*sizeof(char));

  hipMallocManaged((void **)&contig_lens,num_vert*sizeof(uint64_t));




  //setup! want to do parent_cond hook, and then complete iteratively
  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);

  hipDeviceSynchronize();


  auto contig_setup = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> diff = contig_setup-start;

  std::cout << "Time required for contig setup: " << diff.count() << " s\n";

  //init_contigs<<<blocknums, 1024>>>(nnz,num_vert, Arows, Acols, Avals, contigs, contig_lens);

  uint64_t * parents;

  hipMalloc((void **)&parents,num_vert*sizeof(uint64_t));

  init_parent<<<block_vert, 1024>>>(num_vert, parents);

  bool * stars;

  hipMalloc((void ** )&stars, num_vert*sizeof(bool));

  reset_star<<<block_vert, 1024>>>(num_vert, stars);


  simple_adj_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars, contigs, contig_lens);

  hipDeviceSynchronize();

  //now that the device is ready, launch iterative solver
  cuda_solver_kernel<<<block_vert, 1024>>>(nnz, num_vert, contigs, contig_lens, parents, outNnz, kmerVals, kmerLens, kmerParents);

  hipDeviceSynchronize();
  fflush(stdout);

  std::ofstream fout;
  fout.open("iterative_output.dat");

  for (int i = 0; i < outputRows.size(); i++){

    uint64_t row = outputRows.at(i);
    cout << "len: " << contig_lens[row];

    if (contig_lens[row] >= MAX_VEC){
      cout << " TOO LARGE";
    }
    cout <<  endl;
    for (uint64_t j = 0; j < contig_lens[row]; j++){
      fout << contigs[row*MAX_VEC+j];
    }
    fout << endl;

  }
  fout.close();


}



void cc_len(uint64_t nnz, uint64_t num_vert, uint64_t* Arows, uint64_t* Acols, char* Avals, std::vector<uint64_t> outputRows, uint64_t maxOut, char*kmerVals, uint64_t*kmerLens, uint64_t * kmerParents){

  uint64_t blocknums = (nnz -1)/1024 + 1;
  uint64_t block_vert = (num_vert -1)/1024 + 1;


  //move from adj matrix to forward extension per
  char * contigs;
  uint64_t * contig_index;

  auto start = std::chrono::high_resolution_clock::now();

  uint64_t * contig_index_holder;

  uint64_t * contig_lens;

  hipMallocManaged((void **)&contig_index_holder,num_vert*sizeof(uint64_t));

  hipMallocManaged((void **)&contigs,num_vert*sizeof(char));
  hipMallocManaged((void **)&contig_index,num_vert*sizeof(uint64_t));

  hipMallocManaged((void **)&contig_lens,num_vert*sizeof(uint64_t));

  uint64_t * parents;

  hipMalloc((void **)&parents,num_vert*sizeof(uint64_t));

  uint64_t * parents_holder;

  hipMalloc((void **)&parents_holder,num_vert*sizeof(uint64_t));

  init_parent<<<block_vert, 1024>>>(num_vert, parents);

  uint_copy_kernel<<<block_vert, 1024>>>(parents_holder, parents, num_vert);

  bool * stars;

  hipMalloc((void ** )&stars, num_vert*sizeof(bool));


  reset_star<<<block_vert, 1024>>>(num_vert, stars);

  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_index);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_index);

  init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_index_holder);
  assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_index_holder);

  // init_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);
  // assert_contig_lens<<<block_vert, 1024>>>(num_vert, contig_lens);

  hipDeviceSynchronize();

  len_adj_hook<<<blocknums, 1024>>>(nnz, Arows, Acols, Avals, parents, stars, contigs, contig_index);

  //copy over from index at step 1 - these are the correct output lens of the items
  uint_copy_kernel<<<block_vert, 1024>>>(contig_lens, contig_index, num_vert);

  hipDeviceSynchronize();

  //prints
  count_bases<<<1,1>>>(num_vert, contigs);
  sum_lens<<<1,1>>>(num_vert, contig_index);

  hipDeviceSynchronize();
  printf("and lens\n");
  sum_lens<<<1,1>>>(num_vert, contig_lens);
  hipDeviceSynchronize();

  auto full_setup = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> diff = full_setup-start;

  std::cout << "Time required for internal setup: " << diff.count() << " s" << endl;

  fflush(stdout);

  //no errors yet! time to start iterating
  //there is no lead update this time :D

  bool converged = false;
  uint64_t iters = 0;

  do {

    auto iter_start = std::chrono::high_resolution_clock::now();


    //first up is the cond hook
    len_cond_hook<<<block_vert, 1024>>>(num_vert, parents, parents_holder, stars, contig_index, contig_index_holder);

    //then swap pointers

    uint64_t * temp_indices = contig_index;
    contig_index = contig_index_holder;
    contig_index_holder = temp_indices;

    uint64_t * temp_parents = parents;
    parents =  parents_holder;
    parents_holder  = temp_parents;

    //and check stars
    parent_star_check(num_vert, parents, stars);

    converged = starConverged(num_vert, stars);

    hipDeviceSynchronize();

    auto iter_end = std::chrono::high_resolution_clock::now();

    diff = iter_end-iter_start;

    std::cout << "Time required for  iter " << iters << ": " << diff.count() << " s" << endl;

    iters++;

  } while (!converged);

  auto fill_start = std::chrono::high_resolution_clock::now();

  //now that we've converged, we need to map from contig_id to actual values
  uint64_t * contig_map;
  uint64_t * contig_map_lens;

  hipMalloc((void **)&contig_map,maxOut*sizeof(uint64_t));
  hipMalloc((void **)&contig_map_lens,maxOut*sizeof(uint64_t));

  printf("Max output: %llu\n", maxOut);

  uint64_t maxOutBlock = (maxOut -1)/1024 + 1;

  map_contigs<<<maxOutBlock, 1024>>>(maxOut, kmerVals, kmerLens, kmerParents, contig_index, parents, contig_map, contig_map_lens);
  //print_mappings<<<1,1>>>(maxOut, contig_map, contig_map_lens);

  hipDeviceSynchronize();
  fflush(stdout);


  //now that we have the mappings, allocate memory
  char ** final_contigs;
  hipMalloc((void **)&final_contigs,maxOut*sizeof(char * ));

  //char ** host_final_contigs;

  //hipMalloc((void **)&host_final_contigs, maxOut*sizeof(char *));

  mallocHostContigs(maxOut, final_contigs, contig_map_lens);

  //mallocHostContigs(maxOut, host_final_contigs, contig_map_lens);

  //now fill contigs
  fill_contigs_starts<<<maxOutBlock, 1024>>>(maxOut, final_contigs, kmerVals, kmerLens);

  //set up dim3

  //x dim - num vert
  uint64_t x_size = 24;
  uint64_t y_size = 24;
  uint64_t fill_x_block = (num_vert -1)/x_size + 1;
  uint64_t fill_y_block = (maxOut - 1)/y_size + 1;

  dim3 blockShape = dim3(x_size, y_size);
  dim3 gridShape = dim3(fill_x_block, fill_y_block);

  printf("Grid shape: (%llu, %llu)\n", gridShape.x, gridShape.y);
  printf("block shape: (%llu, %llu)\n", blockShape.x, blockShape.y);
  fflush(stdout);

  fill_contigs<<<gridShape, blockShape>>>(num_vert, final_contigs, maxOut, contig_map, contig_map_lens, contigs, contig_index, contig_lens, parents);

  hipDeviceSynchronize();
  fflush(stdout);

  // check_contig<<<1,1>>>(0, final_contigs, contig_map_lens);
  // hipDeviceSynchronize();
  // fflush(stdout);

  auto fill_end = std::chrono::high_resolution_clock::now();

  diff = fill_end-fill_start;

  std::cout << "Time required to fill contig buffers: " << diff.count() << " s" << endl;

  //copy over to host
  // move_contigs_to_host<<<maxOutBlock, 1024>>>(maxOut, final_contigs,host_final_contigs, contig_map_lens);
  // hipDeviceSynchronize();

  save_contigs("cc_len.dat", maxOut, final_contigs, contig_map_lens);
  hipDeviceSynchronize();

  auto write_end = std::chrono::high_resolution_clock::now();

  diff = write_end - fill_end;

  std::cout << "Wrote to file in : " << diff.count() << " s" << endl;


  //and at the end free them

  printf("before host Contigs\n");

  freeHostContigs(maxOut, final_contigs);
  hipDeviceSynchronize();
  printf("Error not in cc_len");
  fflush(stdout);

}









__host__ int cudaMain(int argc, char** argv){

  //start timing
  auto start = std::chrono::high_resolution_clock::now();

  // testAtomic();
  // hipDeviceSynchronize();
  // fflush(stdout);


  auto afterCounter = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> diff = afterCounter-start;


  std::cout << "Tested Counter in " << diff.count() << " s\n";



  //prep matrix info
  std::string kmer_fname = std::string(argv[1]);
  int ks = kmer_size(kmer_fname);

  if (ks != KMER_LEN) {
      throw std::runtime_error("Error: " + kmer_fname + " contains " + std::to_string(ks) +
                               "-mers, while this binary is compiled for " +
                               std::to_string(KMER_LEN) +
                               "-mers.  Modify packing.hpp and recompile.");
  }

    size_t n_kmers = line_count(kmer_fname);


    std::map<string, size_t> map;

    //additional map to assert that the kmer pairs aren't overlapping
    std::map<uint64_t, kmer_pair> err_map;

    //load kmers
    std::vector<kmer_pair> kmers = read_kmers(kmer_fname);

    //host pkmer setup can run asynchronously
    thrust::host_vector<kmer_pair> host_kmers(kmers);


    thrust::host_vector<pkmer_t> host_pkmers(host_kmers.size());

    kmer_pair * host_kmers_ptr = thrust::raw_pointer_cast( host_kmers.data() );

    thrust::transform(host_kmers.begin(), host_kmers.end(), host_pkmers.begin(), kmer_to_pkmer());

    thrust::device_vector<pkmer_t> next_pkmers(host_pkmers);

    pkmer_t * host_pkmers_ptr = thrust::raw_pointer_cast( host_pkmers.data() );

    pkmer_t * dev_pkmers = thrust::raw_pointer_cast( next_pkmers.data() );

    printf("host_pkmers_ptr %p  host_kmers %p\n", host_pkmers_ptr, host_kmers_ptr);

    thrust::device_vector<kmer_pair> dev_kmers_vector(kmers);

    kmer_pair* dev_kmers = thrust::raw_pointer_cast( dev_kmers_vector.data() );


    auto beforeHash = std::chrono::high_resolution_clock::now();

    cudaHashMap * hashMap;

    hashMap = initMap(dev_kmers_vector.size());

    printHashMap(hashMap);

    uint64_t insert_block = (dev_kmers_vector.size() -1)/1024 + 1;

    //original size:  dev_kmers_vector.size()
    insert_all<<<insert_block, 1024>>>(dev_kmers_vector.size(), dev_kmers, hashMap);
    hipDeviceSynchronize();

    printf("Insert completed\n");
    fflush(stdout);


    printHashMap(hashMap);


    //and test correctness
    assertInserts<<<insert_block, 1024>>>(dev_kmers_vector.size(), dev_kmers, hashMap);

    
    auto afterHash = std::chrono::high_resolution_clock::now();

    diff = afterHash-beforeHash;
    std::cout << "Hash table inserts completed in " << diff.count() << " s\n";


    uint64_t * parents;
    hipMalloc((void **)& parents, dev_kmers_vector.size()*sizeof(uint64_t));

    char * extensions;
    hipMalloc((void **)&extensions, dev_kmers_vector.size()*sizeof(char));

    uint64_t num_starts = prep_parents(dev_kmers_vector.size(), dev_kmers, dev_pkmers, parents, extensions, hashMap);

    printf("Num starts: %llu\n", num_starts);

    
    //get kmer leads
    //this is more unavoidable overhead

    thrust::host_vector<pkmer_t> host_leads(host_kmers.size());
    thrust::transform(host_kmers.begin(), host_kmers.end(), host_leads.begin(), kmer_to_start());
    thrust::device_vector<pkmer_t> leads(host_leads);

 
    pkmer_t* lead_pointer = thrust::raw_pointer_cast( leads.data() );




    //now construct starts
    uint64_t startNnz = num_starts;
    char * startVals;
    uint64_t * startLens;
    uint64_t * startRows;
    uint64_t * startIds;

    //locate the starts via cuda, then move to memory based on 
    //find_starts_cuda(dev_kmers_vector.size(), lead_pointer, startNnz, &startIds, hashMap);


    freeCudaHashMap(hashMap);
    return 0;
    
    //everything after this is good but doesn't matter 

    char* perfvals;
    uint64_t * perfrows;
    uint64_t * perfcols;
    uint64_t perf_nnz;

    std::vector<std::pair<kmer_pair, uint64_t>> perf_starts = build_adj_mat(kmers, &perf_nnz, &perfvals, &perfrows, &perfcols);

    //print out some samples

    // for (int i =0; i < 10; i++){
    //   printf("%llu -> %llu: %c\n", perfrows[i], perfcols[i], perfvals[i]);
    // }

    //now run test
    //this is successful for all runtimes
    //build_kmers_from_adj(perf_starts, perf_nnz, perfvals, perfrows, perfcols);


    

    //fill starts mats for cuda
    prep_starts(perf_starts, perfrows, &startNnz, &startVals, &startLens, &startRows);

    //check output for verify
    //on test case looks good
    // printf("Visual sanity check on starts\n");
    // int min_size = 10;
    // if (perf_starts.size() < min_size){
    //   min_size = perf_starts.size();
    // }
    // for (int i=0; i < min_size; i++){
    //   cout << i << ": " << std::get<0>(perf_starts.at(i)).kmer_str() << endl;
    //
    //   cout << i << ": ";
    //   for (int j = 0; j < 10; j++){
    //     cout << startVals[i*MAX_VEC+j];
    //   }
    //   cout << endl;
    // }

    std::vector<uint64_t> outRows2 = gen_outRows(perf_starts, perfrows);

    //what info needs to be updated for the next pass?
    //sync just in case
    hipDeviceSynchronize();


    char* perfvalsCuda;
    uint64_t * perfrowsCuda;
    uint64_t * perfcolsCuda;

    copy_to_cuda(perf_nnz, perfvals, perfrows, perfcols, &perfvalsCuda, &perfrowsCuda, &perfcolsCuda);

    auto midpoint = std::chrono::high_resolution_clock::now();

    diff = midpoint-start;

    std::cout << "Time required for setup: " << diff.count() << " s\n";

    //connected components call
    //iterative_cuda_solver(perf_nnz, n_kmers, perfrowsCuda, perfcolsCuda, perfvalsCuda, outRows2, startNnz, startVals, startLens, startRows);
    cc_len(perf_nnz, n_kmers, perfrowsCuda, perfcolsCuda, perfvalsCuda, outRows2, startNnz, startVals, startLens, startRows);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    diff = end-midpoint;

    std::cout << "Time required for cc: " << diff.count() << " s\n";


    return 0;
}
